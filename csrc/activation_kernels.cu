#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace vllm {

//silu_and_mul_kernel
template <typename T> __device__ __forceinline__ T silu(const T &x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

  
template<typename scalar_t>
__global__ void silu_and_mul_kernel(
  scalar_t* __restrict__ out,               // [..., d]
  const scalar_t* __restrict__ input,       // [..., 2 * d]
  const int d) {

  const int token_idx = blockIdx.x;
  const int64_t token_idx_d = token_idx * int64_t(d);
  const int64_t token_idx_2d = token_idx_d * 2;
  for (int idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = __ldg(&input[token_idx_2d + idx]);
    const scalar_t y = __ldg(&input[token_idx_2d + d + idx]);
    out[token_idx_d + idx] = silu(x) * y;
  }
}

// Activation and gating kernel template.
template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void act_and_mul_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., 2, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * 2 * d + idx]);
    const scalar_t y = VLLM_LDG(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = ACT_FN(x) * y;
  }
}

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

template <typename T>
__device__ __forceinline__ T gelu_kernel(const T& x) {
  // Equivalent to PyTorch GELU with 'none' approximation.
  // Refer to:
  // https://github.com/pytorch/pytorch/blob/8ac9b20d4b090c213799e81acf48a55ea8d437d6/aten/src/ATen/native/cuda/ActivationGeluKernel.cu#L36-L38
  const float f = (float)x;
  constexpr float ALPHA = M_SQRT1_2;
  return (T)(f * 0.5f * (1.0f + ::erf(f * ALPHA)));
}

template <typename T>
__device__ __forceinline__ T gelu_tanh_kernel(const T& x) {
  // Equivalent to PyTorch GELU with 'tanh' approximation.
  // Refer to:
  // https://github.com/pytorch/pytorch/blob/8ac9b20d4b090c213799e81acf48a55ea8d437d6/aten/src/ATen/native/cuda/ActivationGeluKernel.cu#L25-L30
  const float f = (float)x;
  constexpr float BETA = M_SQRT2 * M_2_SQRTPI * 0.5f;
  constexpr float KAPPA = 0.044715;
  float x_cube = f * f * f;
  float inner = BETA * (f + KAPPA * x_cube);
  return (T)(0.5f * f * (1.0f + ::tanhf(inner)));
}

}  // namespace vllm

// Launch activation and gating kernel.
#define LAUNCH_ACTIVATION_GATE_KERNEL(KERNEL)                            \
  int d = input.size(-1) / 2;                                            \
  int64_t num_tokens = input.numel() / input.size(-1);                   \
  dim3 grid(num_tokens);                                                 \
  dim3 block(std::min(d, 1024));                                         \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));      \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();          \
  VLLM_DISPATCH_FLOATING_TYPES(                                          \
      input.scalar_type(), "act_and_mul_kernel", [&] {                   \
        vllm::act_and_mul_kernel<scalar_t, KERNEL<scalar_t>>             \
            <<<grid, block, 0, stream>>>(out.data_ptr<scalar_t>(),       \
                                         input.data_ptr<scalar_t>(), d); \
      });


// from qerve ativation_kernels.cu
// dequant int32 input, apply silu and mul, then per token quant to int8
template <typename scale_type, bool use_per_token_quant>
__global__ void dequant_silu_and_mul_quant_kernel(
    int8_t *__restrict__ out,          // [..., d]
    const int32_t *__restrict__ input, // [..., 2 * d]
    const int d, const float scale_gate, const float scale_up,
    scale_type scale_out,                  // [num_tokens]
    float *__restrict__ tmp = nullptr // [num_tokens, d]
) {
  const int token_idx = blockIdx.x;
  if constexpr (use_per_token_quant) {
    float amax_val = 0.0f;
    const float zero = 0.0f;

    for (int idx = threadIdx.x; idx < d; idx += blockDim.x) {
      const float x =
          (float)__ldg(&input[token_idx * 2 * d + idx]) * scale_gate;
      const float y =
          (float)__ldg(&input[token_idx * 2 * d + d + idx]) * scale_up;
      float t = silu(x) * y;
      tmp[token_idx * d + idx] = t;
      t = t > zero ? t : -t;
      if (t > amax_val)
        amax_val = t;
    }

    __shared__ float s_amax;
    const float block_amax_val = blockReduceMax(amax_val);
    if (threadIdx.x == 0) {
      s_amax = block_amax_val;
      scale_out[token_idx] = block_amax_val / 127.0f;
    }
    __syncthreads();

    float tmp_scale = 127.0f / s_amax;
    for (int idx = threadIdx.x; idx < d; idx += blockDim.x) {
      out[token_idx * d + idx] =
          float_to_int8_rn(tmp_scale * tmp[token_idx * d + idx]);
    }
  } else {
    for (int idx = threadIdx.x; idx < d; idx += blockDim.x) {
      const float x =
          (float)__ldg(&input[token_idx * 2 * d + idx]) * scale_gate;
      const float y =
          (float)__ldg(&input[token_idx * 2 * d + d + idx]) * scale_up;
      out[token_idx * d + idx] = float_to_int8_rn(silu(x) * y / scale_out);
    }
  }
}
} // namespace vllm

void silu_and_mul(torch::Tensor& out,    // [..., d]
                  torch::Tensor& input)  // [..., 2 * d]
{
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::silu_kernel);
}

void gelu_and_mul(torch::Tensor& out,    // [..., d]
                  torch::Tensor& input)  // [..., 2 * d]
{
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::gelu_kernel);
}

void gelu_tanh_and_mul(torch::Tensor& out,    // [..., d]
                       torch::Tensor& input)  // [..., 2 * d]
{
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::gelu_tanh_kernel);
}


// from qerve activation_kernels.cu
void silu_and_mul_qerve(
  torch::Tensor& out,      // [..., d]
  torch::Tensor& input)    // [..., 2 * d]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int d = input.size(-1) / 2;
  dim3 grid(num_tokens);
  dim3 block(std::min(d, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "silu_and_mul_kernel", [&] {
    vllm::silu_and_mul_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), d);
  });
}

void invoke_dequant_silu_and_mul_quant(
    torch::Tensor &out,   // [..., d]
    torch::Tensor &input, // [..., 2 * d]
    const float scale_gate, const float scale_up, const float scale_out) {
  int64_t num_tokens = input.numel() / input.size(-1);
  int d = input.size(-1) / 2;
  dim3 grid(num_tokens);
  dim3 block(std::min(d, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  vllm::dequant_silu_and_mul_quant_kernel<float, false><<<grid, block, 0, stream>>>(
      out.data_ptr<int8_t>(), input.data_ptr<int32_t>(), d, scale_gate,
      scale_up, scale_out);
}


void invoke_dequant_silu_and_mul_quant(
    torch::Tensor &out,   // [..., d]
    torch::Tensor &input, // [..., 2 * d]
    const float scale_gate, const float scale_up,
    torch::Tensor &scale_out, // [num_tokens]
    torch::Tensor &tmp // [..., d]
) {
  int64_t num_tokens = input.numel() / input.size(-1);
  int d = input.size(-1) / 2;
  dim3 grid(num_tokens);
  dim3 block(std::min(d, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  vllm::dequant_silu_and_mul_quant_kernel<float*, true><<<grid, block, 0, stream>>>(
      out.data_ptr<int8_t>(), input.data_ptr<int32_t>(),
       d, scale_gate, scale_up, scale_out.data_ptr<float>(), tmp.data_ptr<float>());
}


namespace vllm {

// Element-wise activation kernel template.
template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void activation_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * d + idx]);
    out[token_idx * d + idx] = ACT_FN(x);
  }
}

}  // namespace vllm

// Launch element-wise activation kernel.
#define LAUNCH_ACTIVATION_KERNEL(KERNEL)                                       \
  int d = input.size(-1);                                                      \
  int64_t num_tokens = input.numel() / d;                                      \
  dim3 grid(num_tokens);                                                       \
  dim3 block(std::min(d, 1024));                                               \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));            \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();                \
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "activation_kernel", [&] { \
    vllm::activation_kernel<scalar_t, KERNEL<scalar_t>>                        \
        <<<grid, block, 0, stream>>>(out.data_ptr<scalar_t>(),                 \
                                     input.data_ptr<scalar_t>(), d);           \
  });

namespace vllm {

template <typename T>
__device__ __forceinline__ T gelu_new_kernel(const T& x) {
  const float x3 = (float)(x * x * x);
  const T t = (T)tanhf((T)(0.79788456f * (float)(x + (T)(0.044715f * x3))));
  return ((T)0.5) * x * (((T)1.0) + t);
}

template <typename T>
__device__ __forceinline__ T gelu_fast_kernel(const T& x) {
  const float f = (float)x;
  const T t =
      (T)tanhf(((T)(f * 0.79788456f)) * (((T)1.0) + (T)(0.044715f * f) * x));
  return ((T)0.5) * x * (((T)1.0) + t);
}

template <typename T>
__device__ __forceinline__ T gelu_quick_kernel(const T& x) {
  // x * sigmoid(1.702 * x)
  return (T)(((float)x) / (1.0f + expf(-1.702f * (float)x)));
}

}  // namespace vllm

void gelu_new(torch::Tensor& out,    // [..., d]
              torch::Tensor& input)  // [..., d]
{
  LAUNCH_ACTIVATION_KERNEL(vllm::gelu_new_kernel);
}

void gelu_fast(torch::Tensor& out,    // [..., d]
               torch::Tensor& input)  // [..., d]
{
  LAUNCH_ACTIVATION_KERNEL(vllm::gelu_fast_kernel);
}

void gelu_quick(torch::Tensor& out,    // [..., d]
                torch::Tensor& input)  // [..., d]
{
  LAUNCH_ACTIVATION_KERNEL(vllm::gelu_quick_kernel);
}
